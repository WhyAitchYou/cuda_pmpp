
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>

inline unsigned int cdiv(unsigned int a, unsigned int b) {
  return (a + b - 1) / b;
}

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vecAdd(float *A_h, float *B_h, float *C_h, int n) {
  // allocate memory on device to init *_d variables
  int size = n * sizeof(float);
  float *A_d, *B_d, *C_d;
  hipMalloc((void **)&A_d, size);
  hipMalloc((void **)&B_d, size);
  hipMalloc((void **)&C_d, size);

  if (!A_d || !B_d || !C_d) {
    printf("CUDA memory allocation failed\n");
    exit(1);
  }
  // copy from host _h to device _d
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
  // compute
  vecAddKernel<<<cdiv(n, 1024), 1024>>>(A_d, B_d, C_d, n);
  // copy from device to host
  hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
  // free allocated memory on device
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  //   float A_h[] = {1, 2, 3};
  //   float B_h[] = {9, 8, 7};
  //   float C_h[3];
  const int size = 100000;
  float A_h[size];
  float B_h[size];
  float C_h[size];
  for (int i = 0; i < size; i++) {
    A_h[i] = (float)rand() / RAND_MAX;
    B_h[i] = (float)rand() / RAND_MAX;
  }
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);
  vecAdd(A_h, B_h, C_h, size);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, end);
  printf("Kernel execution time: %fms\n", milliseconds);
  hipEventDestroy(start);
  hipEventDestroy(end);
}
